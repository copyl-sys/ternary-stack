#include "hip/hip_runtime.h"
// ttypes.cu - CUDA Optimized T81 Arithmetic
#include "ttypes.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BASE_81 81
#define THREADS_PER_BLOCK 256

// **CUDA Kernel: BigInt Addition**
__global__ void t81bigint_add_cuda(int* a, int* b, int* result, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        result[idx] = (a[idx] + b[idx]) % BASE_81;
    }
}

// **CUDA Kernel: BigInt Multiplication**
__global__ void t81bigint_mul_cuda(int* a, int* b, int* result, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        result[idx] = (a[idx] * b[idx]) % BASE_81;
    }
}

// **CUDA Kernel: Matrix Multiplication**
__global__ void matrix_mult_cuda(int* A, int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; k++) {
            sum += (A[row * N + k] * B[k * N + col]) % BASE_81;
        }
        C[row * N + col] = sum % BASE_81;
    }
}

// **CUDA Wrapper for BigInt Add**
void t81bigint_add_gpu(int* h_a, int* h_b, int* h_result, int n) {
    int *d_a, *d_b, *d_result;
    size_t size = n * sizeof(int);
    
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_result, size);
    
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    t81bigint_add_cuda<<<blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_result, n);
    
    hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}

// **CUDA Wrapper for Matrix Multiplication**
void t81matrix_multiply_gpu(int* h_A, int* h_B, int* h_C, int N) {
    int *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(int);
    
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);
    matrix_mult_cuda<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// **Main Function for Testing**
int main() {
    printf("CUDA T81 Ternary System - GPU Accelerated\n");

    // Test BigInt Add
    int h_a[4] = {10, 20, 30, 40};
    int h_b[4] = {5, 15, 25, 35};
    int h_result[4];
    
    t81bigint_add_gpu(h_a, h_b, h_result, 4);
    printf("BigInt Add (GPU): %d %d %d %d\n", h_result[0], h_result[1], h_result[2], h_result[3]);

    // Test Matrix Multiplication
    int N = 4;
    int h_A[16] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    int h_B[16] = {16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
    int h_C[16];

    t81matrix_multiply_gpu(h_A, h_B, h_C, N);
    printf("Matrix Multiply (GPU): %d %d %d %d ...\n", h_C[0], h_C[1], h_C[2], h_C[3]);

    return 0;
}
